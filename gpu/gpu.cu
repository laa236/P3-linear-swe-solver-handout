#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <stdio.h>
#include <math.h>

#include "../common/common.hpp"
#include "../common/solver.hpp"

/**
 * This is your initialization function! We pass in h0, u0, and v0, which are
 * your initial height, u velocity, and v velocity fields. You should send these
 * grids to the GPU so you can do work on them there, and also these other fields.
 * Here, length and width are the length and width of the domain, and nx and ny are
 * the number of grid points in the x and y directions. H is the height of the water
 * column, g is the acceleration due to gravity, and dt is the time step size.
 * The rank and num_procs variables are unused here, but you will need them
 * when doing the MPI version.
 */
// Device pointers
double *h, *u, *v, *dh, *du, *dv, *dh1, *du1, *dv1, *dh2, *du2, *dv2;

// Simulation parameters
double H, g, dt, dx, dy;
int nx, ny;
int t = 0;

//
int numblocks_x, numblocks_y;

void init(double *h0, double *u0, double *v0, double length_, double width_, int nx_, int ny_, double H_, double g_, double dt_, int rank_, int num_procs_)
{
    // Assign values to simulation parameters
    nx = nx_;
    ny = ny_;
    H = H_;
    g = g_;
    dt = dt_;
    dx = length_ / nx;
    dy = width_ / ny;

    size_t size = nx * ny * sizeof(double);
    size_t size_h = (nx + 1) * (ny + 1) * sizeof(double);
    size_t size_u = (nx + 2) * ny * sizeof(double);
    size_t size_v = nx * (ny + 2) * sizeof(double);

    // Allocate device memory
    hipMalloc((void**)&h, size_h);
    hipMalloc((void**)&u, size_u);
    hipMalloc((void**)&v, size_v);

    hipMalloc((void**)&dh, size);
    hipMalloc((void**)&du, size);
    hipMalloc((void**)&dv, size);

    hipMalloc((void**)&dh1, size);
    hipMalloc((void**)&du1, size);
    hipMalloc((void**)&dv1, size);

    hipMalloc((void**)&dh2, size);
    hipMalloc((void**)&du2, size);
    hipMalloc((void**)&dv2, size);

    // Copy initial data to device
    hipMemcpy(h, h0, size_h, hipMemcpyHostToDevice);
    hipMemcpy(u, u0, size_u, hipMemcpyHostToDevice);
    hipMemcpy(v, v0, size_v, hipMemcpyHostToDevice);

    // Initialize derivative arrays to zero
    hipMemset(dh, 0, size);
    hipMemset(du, 0, size);
    hipMemset(dv, 0, size);

    hipMemset(dh1, 0, size);
    hipMemset(du1, 0, size);
    hipMemset(dv1, 0, size);

    hipMemset(dh2, 0, size);
    hipMemset(du2, 0, size);
    hipMemset(dv2, 0, size);

    numblocks_x = (nx + 31) / 32;
    numblocks_y = (ny + 31) / 32;

}

__global__ void ghost_setup(int nx, int ny, double* h) {
    int i = blockIdx.x * 32 + threadIdx.x;
    int j = blockIdx.y * 32 + threadIdx.y;
    //bottom two will only execute on the edges
    //set the top boundary to equal the bottom
    if (i < nx && j == ny) {
        h(i, ny) = h(i, 0);
    }
    //set the right boundary to equal the left
    if (i == nx && j < ny) {
        h(nx, j) = h(0, j);
    }
}

__global__ void calc_derivs(int nx, int ny, double* dh, double* du, double* dv, double* h, double* u, double* v, double H, double g, double dx, double dy) {
    int i = blockIdx.x * 32 + threadIdx.x;
    int j = blockIdx.y * 32 + threadIdx.y;

    if (i >= nx || j >= ny) {
        return;
    }

    dh(i, j) = -H * (du_dx(i, j) + dv_dy(i, j));
    du(i, j) = -g * dh_dx(i, j);
    dv(i, j) = -g * dh_dy(i, j);
}

__global__ void multistep(int nx, int ny, double a1, double a2, double a3, double* dh, double* du, double* dv, double* h, double* u, double* v,
    double* dh1, double* du1, double* dv1, double* dh2, double* du2, double* dv2, double dt)
{
    int i = blockIdx.x * 32 + threadIdx.x;
    int j = blockIdx.y * 32 + threadIdx.y;
    
    if (i >= nx || j >= ny) {
        return;
    }

    h(i, j) += (a1 * dh(i, j) + a2 * dh1(i, j) + a3 * dh2(i, j)) * dt;
    u(i + 1, j) += (a1 * du(i, j) + a2 * du1(i, j) + a3 * du2(i, j)) * dt;
    v(i, j + 1) += (a1 * dv(i, j) + a2 * dv1(i, j) + a3 * dv2(i, j)) * dt;
}

__global__ void compute_boundary(int nx, int ny, double* h, double* u, double* v) {
    int i = blockIdx.x * 32 + threadIdx.x;
    int j = blockIdx.y * 32 + threadIdx.y;
    if (i < nx && j == ny) {
        v(i, 0) = v(i, ny);
    }
    if (i == nx && j < ny) {
        u(0, j) = u(nx, j);
    }
}

void swap_buffers()
{
    double *tmp;

    tmp = dh2;
    dh2 = dh1;
    dh1 = dh;
    dh = tmp;

    tmp = du2;
    du2 = du1;
    du1 = du;
    du = tmp;

    tmp = dv2;
    dv2 = dv1;
    dv1 = dv;
    dv = tmp;
}

/**
 * This is your step function! Here, you will actually numerically solve the shallow
 * water equations. You should update the h, u, and v fields to be the solution after
 * one time step has passed.
 */
void step()
{
    //cuda apparently synchs between kernel calls so no need for the synchs

    /*
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    printf("Max block dimensions: %d x %d x %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    */

    //this block is max threads in a block
    dim3 blockDim(32, 32);
    dim3 gridDim(numblocks_x, numblocks_y);
    
    ghost_setup<<<gridDim, blockDim>>>(nx, ny, h);
    //hipDeviceSynchronize();
    calc_derivs<<<gridDim, blockDim>>>(nx, ny, dh, du, dv, h, u, v, H, g, dx, dy);
    //hipDeviceSynchronize();
    
    double a1, a2, a3;
    if (t == 0)
    {
        a1 = 1.0;
    }
    else if (t == 1)
    {
        a1 = 3.0 / 2.0;
        a2 = -1.0 / 2.0;
    }
    else
    {
        a1 = 23.0 / 12.0;
        a2 = -16.0 / 12.0;
        a3 = 5.0 / 12.0;
    }

    multistep<<<gridDim, blockDim>>>(
        nx, ny, a1, a2, a3, dh, du, dv, h, u, v,
        dh1, du1, dv1, dh2, du2, dv2, dt);
    compute_boundary<<<gridDim, blockDim>>>(nx, ny, h, u, v);
    //hipDeviceSynchronize();
    swap_buffers();
    t++;
}

/**
 * This is your transfer function! You should copy the h field back to the host
 * so that the CPU can check the results of your computation.
 */
void transfer(double *h_host)
{
    size_t size = (nx + 1) * (ny + 1) * sizeof(double);
    hipMemcpy(h_host, h, size, hipMemcpyDeviceToHost);
}

/**
 * This is your finalization function! You should free all of the memory that you
 * allocated on the GPU here.
 */
void free_memory()
{
    hipFree(h);
    hipFree(u);
    hipFree(v);

    hipFree(dh);
    hipFree(du);
    hipFree(dv);

    hipFree(dh1);
    hipFree(du1);
    hipFree(dv1);

    hipFree(dh2);
    hipFree(du2);
    hipFree(dv2);
}